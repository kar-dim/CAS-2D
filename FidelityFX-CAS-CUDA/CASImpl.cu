﻿#include "CAS.cuh"
#include "CASImpl.cuh"
#include "cuda_utils.hpp"
#include <hip/hip_runtime.h>

//initialize empty CAS instance
CASImpl::CASImpl() : texObj(0), texArray(nullptr), casOutputBuffer(nullptr), hostOutputBuffer(nullptr), hasAlpha(false), rows(0), cols(0)
{ }

//destructor, destroy everything
CASImpl::~CASImpl()
{
	destroyBuffers();
}

//initialize buffers and texture data based on the provided image dimensions
void CASImpl::initializeMemory()
{
	const int channels = hasAlpha ? 4 : 3;
	//initialize CAS output buffers and pinned memory for output
	hipMalloc(&casOutputBuffer, sizeof(unsigned char) * channels * rows * cols);
	hipHostAlloc((void**)&hostOutputBuffer, sizeof(unsigned char) * channels * rows * cols, hipHostMallocDefault);
	//initialize texture
	auto textureData = cuda_utils::createTextureData(rows, cols);
	texObj = textureData.first;
	texArray = textureData.second;
}

//destory and re-initialize memory objects
void CASImpl::reinitializeMemory(const bool hasAlpha, const unsigned char* hostRgbPtr, const unsigned int rows, const unsigned int cols)
{
	this->rows = rows;
	this->cols = cols;
	this->hasAlpha = hasAlpha;
	destroyBuffers();
	initializeMemory();
	cuda_utils::copyDataToCudaArray(hostRgbPtr, rows, cols, texArray);
}

//delete all buffers
void CASImpl::destroyBuffers()
{
	static constexpr auto destroy = [](auto& resource, auto& deleter) { if (resource) deleter(resource); };
	destroy(casOutputBuffer, hipFree);
	destroy(texObj, hipDestroyTextureObject);
	destroy(texArray, hipFreeArray);
	destroy(hostOutputBuffer, hipHostFree);
}

//calls CAS kernel on the texture data, return sharpened image as unsigned char buffer (pinned memory of this CAS instance)
//overloaded method to be used when the texture data is already set (get away with one Host to Device copy if we want to sharpen the same image)
const unsigned char* CASImpl::sharpenImage(const int casMode, const float sharpenStrength, const float contrastAdaption)
{
	const dim3 gridSize = cuda_utils::gridSizeCalculate(blockSize, rows, cols);
	//enqueue CAS kernel with Alpha channel output or not, or RGB planar or interleaved output based on param casMode
	if (hasAlpha && casMode == PLANAR_RGB)
		cas <true, PLANAR_RGB> << <gridSize, blockSize >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
	else if (hasAlpha && casMode == INTERLEAVED_RGBA)
		cas <true, INTERLEAVED_RGBA> << <gridSize, blockSize >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
	else if (!hasAlpha && casMode == PLANAR_RGB)
		cas <false, PLANAR_RGB> << <gridSize, blockSize >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
	else
		cas <false, INTERLEAVED_RGBA> << <gridSize, blockSize >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);

	//copy from GPU to HOST
	hipMemcpy(hostOutputBuffer, casOutputBuffer, rows * cols * sizeof(unsigned char) * (hasAlpha ? 4 : 3), hipMemcpyDeviceToHost);
	return hostOutputBuffer;
}
