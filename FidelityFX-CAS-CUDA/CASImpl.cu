﻿#include "CAS.cuh"
#include "CASImpl.cuh"
#include "cuda_utils.hpp"
#include <hip/hip_runtime.h>

void CASImpl::initializeMemory(const unsigned int rows, const unsigned int cols)
{
	//initialize CAS output buffers and pinned memory for output
	hipMallocAsync(&casOutputBufferRGB, sizeof(unsigned char) * rows * cols * 3, stream);
	hipMallocAsync(&casOutputBufferR, sizeof(unsigned char) * rows * cols, streamR);
	hipMallocAsync(&casOutputBufferG, sizeof(unsigned char) * rows * cols, streamG);
	hipMallocAsync(&casOutputBufferB, sizeof(unsigned char) * rows * cols, streamB);
	hipHostAlloc((void**)&hostOutputBuffer, sizeof(unsigned char) * rows * cols * 3, hipHostMallocDefault);
	cuda_utils::cudaStreamsSynchronize(stream,streamR, streamG, streamB);
	//initialize texture
	auto textureData = cuda_utils::createTextureData(rows,cols);
	texObj = textureData.first;
	texArray = textureData.second;
}

//full constructor
CASImpl::CASImpl(const unsigned int rows, const unsigned int cols) :
	rows(rows), cols(cols)
{
	cuda_utils::cudaStreamsCreate(stream, streamR, streamG, streamB);
	initializeMemory(rows, cols);
}

//copy constructor
CASImpl::CASImpl(const CASImpl& other):
	rows(other.rows), cols(other.rows)
{
	cuda_utils::cudaStreamsCreate(stream, streamR, streamG, streamB);
	initializeMemory(rows, cols);
}

//move constructor
CASImpl::CASImpl(CASImpl&& other) noexcept :
	rows(other.rows), cols(other.rows)
{
	//move buffers and texture data and nullify other
	casOutputBufferRGB = other.casOutputBufferRGB;
	casOutputBufferR = other.casOutputBufferR;
	casOutputBufferG = other.casOutputBufferG;
	casOutputBufferB = other.casOutputBufferB;
	hostOutputBuffer = other.hostOutputBuffer;
	texObj = other.texObj;
	texArray = other.texArray;
	stream = other.stream;
	streamR = other.streamR;
	streamG = other.streamG;
	streamB = other.streamB;
	other.casOutputBufferRGB = nullptr;
	other.casOutputBufferR = nullptr;
	other.casOutputBufferG = nullptr;
	other.casOutputBufferB = nullptr;
	other.hostOutputBuffer = nullptr;
	other.texObj = 0;
	other.stream = nullptr;
	other.streamR = nullptr;
	other.streamG = nullptr;
	other.streamB = nullptr;
	other.texArray = nullptr;
}

//move assignment
CASImpl& CASImpl::operator=(CASImpl&& other) noexcept
{
	if (this != &other)
	{
		rows = other.rows;
		cols = other.cols;
		//move pitched memory
		hipHostFree(hostOutputBuffer);
		hostOutputBuffer = other.hostOutputBuffer;
		other.hostOutputBuffer = nullptr;
		//move streams
		cuda_utils::cudaStreamsDestroy(stream, streamR, streamG, streamB);
		stream = other.stream;
		other.stream = nullptr;
		streamR = other.streamR;
		other.streamR = nullptr;
		streamG = other.streamG;
		other.streamG = nullptr;
		streamB = other.streamB;
		other.streamB = nullptr;
		//move texture object
		hipDestroyTextureObject(texObj);
		texObj = other.texObj;
		other.texObj = 0;
		//move texture array
		hipFreeArray(texArray);
		texArray = other.texArray;
		other.texArray = nullptr;
	}
	return *this;
}

//copy assignment
CASImpl& CASImpl::operator=(const CASImpl& other)
{
	if (this != &other)
	{
		rows = other.rows;
		cols = other.cols;
		hipHostFree(hostOutputBuffer);
		hipDestroyTextureObject(texObj);
		hipFreeArray(texArray);
		initializeMemory(rows, cols);
	}
	return *this;
}

void CASImpl::destroyBuffers()
{
	static constexpr auto destroy = [](auto&& resource, auto&& deleter) { if (resource) deleter(resource); };
	static constexpr auto destroyAsync = [](auto&& resource, auto&& stream, auto&& deleter) { if (resource) deleter(resource, stream); };
	destroyAsync(casOutputBufferRGB, stream, hipFreeAsync);
	destroyAsync(casOutputBufferR, streamR, hipFreeAsync);
	destroyAsync(casOutputBufferG, streamG, hipFreeAsync);
	destroyAsync(casOutputBufferB, streamB, hipFreeAsync);
	cuda_utils::cudaStreamsSynchronize(stream, streamR, streamG, streamB);
	destroy(texObj, hipDestroyTextureObject);
	destroy(texArray, hipFreeArray);
	destroy(hostOutputBuffer, hipHostFree);
}

//destructor, destroy everything
CASImpl::~CASImpl()
{
	destroyBuffers();
	cuda_utils::cudaStreamsDestroy(stream, streamR, streamG, streamB);
}

//destory and re-initialize memory objects only
void CASImpl::reinitializeMemory(const unsigned int rows, const unsigned int cols)
{
	this->rows = rows;
	this->cols = cols;
	destroyBuffers();
	initializeMemory(rows, cols);
}

//setup and call main CAS kernel, return sharpened image as unsigned char buffer (pinned memory of this CAS instance)
//inputImage must be interleaved RGB data
const unsigned char* CASImpl::sharpenImage(const unsigned char *inputImage, const CASMode casMode, const float sharpenStrength, const float contrastAdaption)
{
	const dim3 blockSize(16, 16);
	const dim3 gridSize = cuda_utils::gridSizeCalculate(blockSize, rows, cols);
	//copy input data to texture
	cuda_utils::copyDataToCudaArray(inputImage, rows, cols, texArray, stream);
	
	if (casMode == CASMode::CAS_RGB) 
	{
		//enqueue CAS kernel
		cas<CASMode::CAS_RGB> << <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBufferR, casOutputBufferG, casOutputBufferB, casOutputBufferRGB, rows, cols);
		hipStreamSynchronize(stream);
		//copy from GPU to HOST
		hipMemcpyAsync(hostOutputBuffer, casOutputBufferR, rows * cols * sizeof(unsigned char), hipMemcpyDefault, streamR);
		hipMemcpyAsync(hostOutputBuffer + (rows * cols), casOutputBufferG, rows * cols * sizeof(unsigned char), hipMemcpyDefault, streamG);
		hipMemcpyAsync(hostOutputBuffer + (2 * (rows * cols)), casOutputBufferB, rows * cols * sizeof(unsigned char), hipMemcpyDefault, streamB);
		cuda_utils::cudaStreamsSynchronize(streamR, streamG, streamB);
	}
	else 
	{
		//enqueue CAS kernel
		cas<CASMode::CAS_INTERLEAVED> << <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBufferR, casOutputBufferG, casOutputBufferB, casOutputBufferRGB, rows, cols);
		hipStreamSynchronize(stream);
		//copy from GPU to HOST
		hipMemcpyAsync(hostOutputBuffer, casOutputBufferRGB, rows * cols * sizeof(unsigned char) * 3, hipMemcpyDefault, stream);
		hipStreamSynchronize(stream);
	}
	return hostOutputBuffer;
}
