﻿#include "CAS.cuh"
#include "CASImpl.cuh"
#include "cuda_utils.hpp"
#include <hip/hip_runtime.h>

void CASImpl::initializeMemory(const unsigned int rows, const unsigned int cols)
{
	//initialize CAS output buffers and pinned memory for output
	hipMallocAsync(&casOutputBufferR, sizeof(unsigned char) * rows * cols, streamR);
	hipMallocAsync(&casOutputBufferG, sizeof(unsigned char) * rows * cols, streamG);
	hipMallocAsync(&casOutputBufferB, sizeof(unsigned char) * rows * cols, streamB);
	hipHostAlloc((void**)&pitchedBuffer, sizeof(unsigned char) * rows * cols * 3, hipHostMallocDefault);
	cuda_utils::cudaStreamsSynchronize(streamR, streamG, streamB);
	//initialize texture
	auto textureData = cuda_utils::createTextureData(rows,cols);
	texObj = textureData.first;
	texArray = textureData.second;
}

//full constructor
CASImpl::CASImpl(const unsigned int rows, const unsigned int cols, const float sharpenStrength, const float contrastAdaption):
	sharpenStrength(sharpenStrength), contrastAdaption(contrastAdaption), rows(rows), cols(cols)
{
	cuda_utils::cudaStreamsCreate(stream, streamR, streamG, streamB);
	initializeMemory(rows, cols);
}

//copy constructor
CASImpl::CASImpl(const CASImpl& other):
	sharpenStrength(other.sharpenStrength), contrastAdaption(other.contrastAdaption), rows(other.rows), cols(other.rows)
{
	cuda_utils::cudaStreamsCreate(stream, streamR, streamG, streamB);
	initializeMemory(rows, cols);
}

//move constructor
CASImpl::CASImpl(CASImpl&& other) noexcept :
	sharpenStrength(other.sharpenStrength), contrastAdaption(other.contrastAdaption), rows(other.rows), cols(other.rows)
{
	//move buffers and texture data and nullify other
	casOutputBufferR = other.casOutputBufferR;
	casOutputBufferG = other.casOutputBufferG;
	casOutputBufferB = other.casOutputBufferB;
	pitchedBuffer = other.pitchedBuffer;
	texObj = other.texObj;
	texArray = other.texArray;
	stream = other.stream;
	streamR = other.streamR;
	streamG = other.streamG;
	streamB = other.streamB;
	other.casOutputBufferR = nullptr;
	other.casOutputBufferG = nullptr;
	other.casOutputBufferB = nullptr;
	other.pitchedBuffer = nullptr;
	other.texObj = 0;
	other.stream = nullptr;
	other.streamR = nullptr;
	other.streamG = nullptr;
	other.streamB = nullptr;
	other.texArray = nullptr;
}

//move assignment
CASImpl& CASImpl::operator=(CASImpl&& other) noexcept
{
	if (this != &other)
	{
		sharpenStrength = other.sharpenStrength;
		contrastAdaption = other.contrastAdaption;
		rows = other.rows;
		cols = other.cols;
		//move pitched memory
		hipHostFree(pitchedBuffer);
		pitchedBuffer = other.pitchedBuffer;
		other.pitchedBuffer = nullptr;
		//move streams
		cuda_utils::cudaStreamsDestroy(stream, streamR, streamG, streamB);
		stream = other.stream;
		other.stream = nullptr;
		streamR = other.streamR;
		other.streamR = nullptr;
		streamG = other.streamG;
		other.streamG = nullptr;
		streamB = other.streamB;
		other.streamB = nullptr;
		//move texture object
		hipDestroyTextureObject(texObj);
		texObj = other.texObj;
		other.texObj = 0;
		//move texture array
		hipFreeArray(texArray);
		texArray = other.texArray;
		other.texArray = nullptr;
	}
	return *this;
}

//copy assignment
CASImpl& CASImpl::operator=(const CASImpl& other)
{
	if (this != &other)
	{
		sharpenStrength = other.sharpenStrength;
		contrastAdaption = other.contrastAdaption;
		rows = other.rows;
		cols = other.cols;
		hipHostFree(pitchedBuffer);
		hipDestroyTextureObject(texObj);
		hipFreeArray(texArray);
		initializeMemory(rows, cols);
	}
	return *this;
}

//destructor
CASImpl::~CASImpl()
{
	static constexpr auto destroy = [](auto&& resource, auto&& deleter) { if (resource) deleter(resource); };
	static constexpr auto destroyAsync = [](auto&& resource, auto &&stream, auto&& deleter) { if (resource) deleter(resource, stream); };
	destroyAsync(casOutputBufferR, streamR, hipFreeAsync);
	destroyAsync(casOutputBufferG, streamG, hipFreeAsync);
	destroyAsync(casOutputBufferB, streamB, hipFreeAsync);
	cuda_utils::cudaStreamsSynchronize(streamR, streamG, streamB);
	cuda_utils::cudaStreamsDestroy(stream, streamR, streamG, streamB);
	destroy(texObj, hipDestroyTextureObject);
	destroy(texArray, hipFreeArray);
	destroy(pitchedBuffer, hipHostFree);
}

//destory and re-initialize memory objects, and setup new parameter values
void CASImpl::reinitialize(const unsigned int rows, const unsigned int cols, const float sharpenStrength, const float contrastAdaption)
{
	this->sharpenStrength = sharpenStrength;
	this->contrastAdaption = contrastAdaption;
	hipDestroyTextureObject(texObj);
	hipFreeArray(texArray);
	hipHostFree(pitchedBuffer);
	initializeMemory(rows, cols);
}

//setup and call main CAS kernel, return sharpened image as unsigned char buffer (pinned memory of this CAS instance)
const unsigned char* CASImpl::sharpenImage(const unsigned char *inputImage)
{
	const dim3 blockSize(16, 16);
	const dim3 gridSize = cuda_utils::gridSizeCalculate(blockSize, rows, cols);
	//copy input data to texture
	cuda_utils::copyDataToCudaArray(inputImage, rows, cols, texArray, stream);
	//enqueue CAS kernel
	cas << <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBufferR, casOutputBufferG, casOutputBufferB, rows, cols);
	hipStreamSynchronize(stream);
	//copy from GPU to HOST
	hipMemcpyAsync(pitchedBuffer, casOutputBufferR, rows * cols * sizeof(unsigned char), hipMemcpyDefault, streamR);
	hipMemcpyAsync(pitchedBuffer + (rows * cols), casOutputBufferG, rows * cols * sizeof(unsigned char), hipMemcpyDefault, streamG);
	hipMemcpyAsync(pitchedBuffer + (2 * (rows * cols)), casOutputBufferB, rows * cols * sizeof(unsigned char), hipMemcpyDefault, streamB);
	cuda_utils::cudaStreamsSynchronize(streamR, streamG, streamB);
	return pitchedBuffer;
}
