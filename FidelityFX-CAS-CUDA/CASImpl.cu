﻿#include "CAS.cuh"
#include "CASImpl.cuh"
#include "cuda_utils.hpp"
#include <hip/hip_runtime.h>
#include <type_traits>

//initialize buffers and texture data based on the provided image dimensions
void CASImpl::initializeMemory()
{
	const int channels = hasAlpha ? 4 : 3;
	//initialize CAS output buffers and pinned memory for output
	hipMallocAsync(&casOutputBuffer, sizeof(unsigned char) * channels * rows * cols, stream);
	hipHostAlloc((void**)&hostOutputBuffer, sizeof(unsigned char) * channels * rows * cols, hipHostMallocDefault);
	cuda_utils::cudaStreamsSynchronize(stream);
	//initialize texture
	auto textureData = cuda_utils::createTextureData(rows, cols);
	texObj = textureData.first;
	texArray = textureData.second;
}

//full constructor
CASImpl::CASImpl(const bool hasAlpha, const unsigned int rows, const unsigned int cols) :
	hasAlpha(hasAlpha), rows(rows), cols(cols)
{
	cuda_utils::cudaStreamsCreate(stream);
	initializeMemory();
}

//copy constructor
CASImpl::CASImpl(const CASImpl& other):
	hasAlpha(other.hasAlpha), rows(other.rows), cols(other.rows)
{
	cuda_utils::cudaStreamsCreate(stream);
	initializeMemory();
}

//helper method for moving data between CAS instances
void CASImpl::moveData(CASImpl&& other) noexcept
{
	static constexpr auto moveMember = [](auto& thisData, auto& otherData) { thisData = otherData; otherData = nullptr; };
	rows = other.rows;
	cols = other.cols;
	hasAlpha = other.hasAlpha;
	//move buffer/pointer data between this and other
	moveMember(hostOutputBuffer, other.hostOutputBuffer);
	moveMember(casOutputBuffer, other.casOutputBuffer);
	moveMember(stream, other.stream);
	moveMember(texArray, other.texArray);
	//move texture object
	texObj = other.texObj;
	other.texObj = 0;
}

//move constructor
CASImpl::CASImpl(CASImpl&& other) noexcept
{
	moveData(std::move(other));
}

//move assignment
CASImpl& CASImpl::operator=(CASImpl&& other) noexcept
{
	if (this != &other)
	{
		//delete old buffers and stream and move data from other to this
		destroyBuffers();
		cuda_utils::cudaStreamsDestroy(stream);
		moveData(std::move(other));
	}
	return *this;
}

//copy assignment
CASImpl& CASImpl::operator=(const CASImpl& other)
{
	if (this != &other) 
	{
		//no need to reinitialize streams, only buffers
		reinitializeMemory(other.hasAlpha, other.rows, other.cols);
	}
	return *this;
}

//delete all buffers
void CASImpl::destroyBuffers()
{
	static constexpr auto destroy = [](auto& resource, auto& deleter) { if (resource) deleter(resource); };
	static constexpr auto destroyAsync = [](auto& resource, auto& stream, auto& deleter) { if (resource) deleter(resource, stream); };
	destroyAsync(casOutputBuffer, stream, hipFreeAsync);
	destroy(texObj, hipDestroyTextureObject);
	destroy(texArray, hipFreeArray);
	destroy(hostOutputBuffer, hipHostFree);
	cuda_utils::cudaStreamsSynchronize(stream);
}

//destructor, destroy everything
CASImpl::~CASImpl()
{
	destroyBuffers();
	cuda_utils::cudaStreamsDestroy(stream);
}

//destory and re-initialize memory objects only
void CASImpl::reinitializeMemory(const bool hasAlpha, const unsigned int rows, const unsigned int cols)
{
	this->rows = rows;
	this->cols = cols;
	this->hasAlpha = hasAlpha;
	destroyBuffers();
	initializeMemory();
}

//setup and call main CAS kernel, return sharpened image as unsigned char buffer (pinned memory of this CAS instance)
//hostRgbPtr must be interleaved RGB(A) data
//returns the sharpened image as unsigned char buffer (planar RGB or interleaved RGBA, based on casMode param)
const unsigned char* CASImpl::sharpenImage(const unsigned char *hostRgbPtr, const int casMode, const float sharpenStrength, const float contrastAdaption)
{
	const dim3 gridSize = cuda_utils::gridSizeCalculate(blockSize, rows, cols);
	//copy input data to texture
	cuda_utils::copyDataToCudaArrayAsync(hostRgbPtr, rows, cols, texArray, stream);
	hipStreamSynchronize(stream);
	//enqueue CAS kernel with Alpha channel output or not, or RGB planar or interleaved output based on param casMode
	if (hasAlpha && casMode == PLANAR_RGB)
		cas <true, PLANAR_RGB> << <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
	else if (hasAlpha && casMode == INTERLEAVED_RGBA)
		cas <true, INTERLEAVED_RGBA> << <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
	else if (!hasAlpha && casMode == PLANAR_RGB)
		cas <false, PLANAR_RGB> << <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
	else
		cas <false, INTERLEAVED_RGBA> << <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
	hipStreamSynchronize(stream);
	//copy from GPU to HOST
	hipMemcpyAsync(hostOutputBuffer, casOutputBuffer, rows * cols * sizeof(unsigned char) * (hasAlpha ? 4 : 3), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	return hostOutputBuffer;
}
