﻿#include "CAS.cuh"
#include "CASImpl.cuh"
#include "cuda_utils.hpp"
#include <hip/hip_runtime.h>

void CASImpl::initializeMemory(const unsigned int rows, const unsigned int cols)
{
	const int channels = hasAlpha ? 4 : 3;
	//initialize CAS output buffers and pinned memory for output
	hipMallocAsync(&casOutputBuffer, sizeof(unsigned char) * channels * rows * cols, stream);
	hipHostAlloc((void**)&hostOutputBuffer, sizeof(unsigned char) * channels * rows * cols, hipHostMallocDefault);
	cuda_utils::cudaStreamsSynchronize(stream);
	//initialize texture
	auto textureData = cuda_utils::createTextureData(rows, cols);
	texObj = textureData.first;
	texArray = textureData.second;
}

//full constructor
CASImpl::CASImpl(const bool hasAlpha, const unsigned int rows, const unsigned int cols) :
	hasAlpha(hasAlpha), rows(rows), cols(cols)
{
	cuda_utils::cudaStreamsCreate(stream);
	initializeMemory(rows, cols);
}

//copy constructor
CASImpl::CASImpl(const CASImpl& other):
	hasAlpha(other.hasAlpha), rows(other.rows), cols(other.rows)
{
	cuda_utils::cudaStreamsCreate(stream);
	initializeMemory(rows, cols);
}

//move constructor
CASImpl::CASImpl(CASImpl&& other) noexcept :
	hasAlpha(other.hasAlpha), rows(other.rows), cols(other.rows)
{
	//move buffers and texture data and nullify other
	casOutputBuffer = other.casOutputBuffer;
	hostOutputBuffer = other.hostOutputBuffer;
	texObj = other.texObj;
	texArray = other.texArray;
	stream = other.stream;
	other.casOutputBuffer = nullptr;
	other.hostOutputBuffer = nullptr;
	other.texObj = 0;
	other.stream = nullptr;
	other.texArray = nullptr;
}

//move assignment
CASImpl& CASImpl::operator=(CASImpl&& other) noexcept
{
	if (this != &other)
	{
		rows = other.rows;
		cols = other.cols;
		hasAlpha = other.hasAlpha;
		//move pitched memory
		hipHostFree(hostOutputBuffer);
		hostOutputBuffer = other.hostOutputBuffer;
		other.hostOutputBuffer = nullptr;
		//move streams
		cuda_utils::cudaStreamsDestroy(stream);
		stream = other.stream;
		other.stream = nullptr;
		//move texture object
		hipDestroyTextureObject(texObj);
		texObj = other.texObj;
		other.texObj = 0;
		//move texture array
		hipFreeArray(texArray);
		texArray = other.texArray;
		other.texArray = nullptr;
	}
	return *this;
}

//copy assignment
CASImpl& CASImpl::operator=(const CASImpl& other)
{
	if (this != &other)
	{
		rows = other.rows;
		cols = other.cols;
		hasAlpha = other.hasAlpha;
		hipHostFree(hostOutputBuffer);
		hipDestroyTextureObject(texObj);
		hipFreeArray(texArray);
		initializeMemory(rows, cols);
	}
	return *this;
}

void CASImpl::destroyBuffers()
{
	static constexpr auto destroy = [](auto&& resource, auto&& deleter) { if (resource) deleter(resource); };
	static constexpr auto destroyAsync = [](auto&& resource, auto&& stream, auto&& deleter) { if (resource) deleter(resource, stream); };
	destroyAsync(casOutputBuffer, stream, hipFreeAsync);
	destroy(texObj, hipDestroyTextureObject);
	destroy(texArray, hipFreeArray);
	destroy(hostOutputBuffer, hipHostFree);
	cuda_utils::cudaStreamsSynchronize(stream);
}

//destructor, destroy everything
CASImpl::~CASImpl()
{
	destroyBuffers();
	cuda_utils::cudaStreamsDestroy(stream);
}

//destory and re-initialize memory objects only
void CASImpl::reinitializeMemory(const bool hasAlpha, const unsigned int rows, const unsigned int cols)
{
	this->rows = rows;
	this->cols = cols;
	this->hasAlpha = hasAlpha;
	destroyBuffers();
	initializeMemory(rows, cols);
}

//setup and call main CAS kernel, return sharpened image as unsigned char buffer (pinned memory of this CAS instance)
//hostRgbPtr must be interleaved RGB(A) data
const unsigned char* CASImpl::sharpenImage(const unsigned char *hostRgbPtr, const float sharpenStrength, const float contrastAdaption)
{
	const dim3 gridSize = cuda_utils::gridSizeCalculate(blockSize, rows, cols);
	//copy input data to texture
	cuda_utils::copyDataToCudaArrayAsync(hostRgbPtr, rows, cols, texArray, stream);
	hipStreamSynchronize(stream);
	if (hasAlpha) 
	{
		//enqueue CAS kernel
		cas <true><< <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
		hipStreamSynchronize(stream);
		//copy from GPU to HOST
		hipMemcpyAsync(hostOutputBuffer, casOutputBuffer, rows * cols * sizeof(unsigned char) * 4, hipMemcpyDeviceToHost, stream);
	}
	else 
	{
		//enqueue CAS kernel
		cas <false><< <gridSize, blockSize, 0, stream >> > (texObj, sharpenStrength, contrastAdaption, casOutputBuffer, rows, cols);
		hipStreamSynchronize(stream);
		//copy from GPU to HOST
		hipMemcpyAsync(hostOutputBuffer, casOutputBuffer, rows * cols * sizeof(unsigned char) * 3, hipMemcpyDeviceToHost, stream);
	}
	hipStreamSynchronize(stream);
	return hostOutputBuffer;
}
